#include "hip/hip_runtime.h"
#include "tsdf.cuh"
#include "cuda_event.h"
#include "device_helper.cuh"

#include <hip/hip_fp16.h>

__global__ void fuseKernel(hipSurfaceObject_t raw_depth_map, VoxelGridStruct voxel_grid, Sensor sensor)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	auto* char_ptr = static_cast<char*>(voxel_grid.pointer.ptr);
	auto* ptr = reinterpret_cast<Voxel*>(char_ptr + voxel_grid.pointer.pitch * (voxel_grid.n * i + j));

	const auto resolution = voxel_grid.resolution;
	const auto mue = voxel_grid.mue;
	auto start_coord = (-voxel_grid.total_width_in_meters + resolution) * 0.5f;
	glm::vec3 point(start_coord + i * resolution, start_coord + j * resolution, start_coord - resolution);
	for (int k = 0; k < voxel_grid.n; ++k)
	{
		//Update the point to the center of the next voxel.
		point.z += resolution;

		//1-Project point into image space of sensor and perform homogeneous division.
		glm::vec3 point_eye = sensor.getInversePose() * glm::vec4(point, 1.0f);
		auto pixel = sensor.getIntr() * point_eye;
		pixel /= pixel.z;

		//2-Check if it is in the view frustum. If not, don't do anything.
		if (!(point_eye.z >= kernel::cMinDistance && point_eye.z < kernel::cMaxDistance &&
			pixel.x >= 0.0f && pixel.x < 640.0f && pixel.y >= 0.0f && pixel.y < 480.0f))
		{
			continue;
		}

		//3-Take depth value by nearest neighbour lookup.
		unsigned short h_depth;
		surf2Dread(&h_depth, raw_depth_map, static_cast<int>(pixel.x) * 2, static_cast<int>(pixel.y));
		auto depth = __half2float(h_depth);

		//If depth value is invalid, continue with the next voxel.
		if (!device_helper::isDepthValid(depth))
		{
			continue;
		}

		auto diff = depth - point_eye.z;
		if (diff >= -mue)
		{
			//4-Compute TSDF and weight;
			auto f = glm::min(1.0f, diff / mue) * glm::sign(diff);
			auto w = 1.0f;

			//5-Update voxel.f and voxel.w
			auto& voxel = ptr[k];
			voxel.f = (voxel.f * voxel.w + f * w) / (voxel.w + w);
			voxel.w += w;
		}
	}
}

namespace kernel
{
	float fuse(hipSurfaceObject_t raw_depth_map, const VoxelGridStruct& voxel_grid, const Sensor& sensor)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(voxel_grid.n / threads.x, voxel_grid.n / threads.y);
		start.record();
		fuseKernel <<<blocks, threads>>> (raw_depth_map, voxel_grid, sensor);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}
}