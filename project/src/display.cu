#include "hip/hip_runtime.h"
#include "display.cuh"
#include "device_helper.cuh"
#include "glm_macro.h"
#include "cuda_event.h"

#include <hip/hip_fp16.h>
#include <glm/vec3.hpp>

__global__ void oneHalfChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned short h_pixel;
	surf2Dread(&h_pixel, surface, i * 2, j, hipBoundaryModeZero);

	auto pixel = static_cast<unsigned char>(__half2float(h_pixel) * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window, i * 4, j);
}

__global__ void oneFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float f_pixel;
	surf2Dread(&f_pixel, surface, i * 4, j, hipBoundaryModeZero);

	auto pixel = static_cast<unsigned char>(f_pixel * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window, i * 4, j);
}

__global__ void fourFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float r, g, b;
	int idx = i * 16;
	surf2Dread(&r, surface, idx, j, hipBoundaryModeZero);
	surf2Dread(&g, surface, idx + 4, j, hipBoundaryModeZero);
	surf2Dread(&b, surface, idx + 8, j, hipBoundaryModeZero);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(b * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(g * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(r * scale));

	surf2Dwrite(pixel_w, window, i * 4, j);
}

__global__ void normalMapToWindowContentKernel(hipSurfaceObject_t normal_map, hipSurfaceObject_t window, glm::mat3 inverse_sensor_rotation)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	glm::vec3 normal;
	int idx = i * 16;
	surf2Dread(&normal.x, normal_map, idx, j, hipBoundaryModeZero);
	surf2Dread(&normal.y, normal_map, idx + 4, j, hipBoundaryModeZero);
	surf2Dread(&normal.z, normal_map, idx + 8, j, hipBoundaryModeZero);

	normal = glm::normalize(inverse_sensor_rotation * normal);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(normal.z * 255.0f)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(normal.y * 255.0f)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(normal.x * 255.0f));

	surf2Dwrite(pixel_w, window, i * 4, j);
}

__global__ void shadingToWindowContentKernel(hipSurfaceObject_t normal_map, hipSurfaceObject_t window, Sensor sensor)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	glm::vec3 normal;
	int idx = i * 16;
	surf2Dread(&normal.x, normal_map, idx, j, hipBoundaryModeZero);
	surf2Dread(&normal.y, normal_map, idx + 4, j, hipBoundaryModeZero);
	surf2Dread(&normal.z, normal_map, idx + 8, j, hipBoundaryModeZero);

	auto ray_direction = glm::normalize(glm::mat3(sensor.getPose()) * sensor.getInverseIntr(0) * glm::vec3(i + 0.5f, j + 0.5f, 1.0f));
	auto radiance = glm::min(glm::abs(glm::dot(normal, ray_direction)), 1.0f);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(radiance * 255.0f)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(radiance * 255.0f)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(radiance * 255.0f));

	surf2Dwrite(pixel_w, window, i * 4, j);
}

namespace kernel
{
	float oneHalfChannelToWindowContent(hipSurfaceObject_t surface, const Window& window, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneHalfChannelToWindowContentKernel << <blocks, threads >> > (surface, window.get_content(), scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float oneFloatChannelToWindowContent(hipSurfaceObject_t surface, const Window& window, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneFloatChannelToWindowContentKernel << <blocks, threads >> > (surface, window.get_content(), scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float fourFloatChannelToWindowContent(hipSurfaceObject_t surface, const Window& window, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		fourFloatChannelToWindowContentKernel << <blocks, threads >> > (surface, window.get_content(), scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float normalMapToWindowContent(hipSurfaceObject_t normal_map, const Window& window, const glm::mat3& inverse_sensor_rotation)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		normalMapToWindowContentKernel << <blocks, threads >> > (normal_map, window.get_content(), inverse_sensor_rotation);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float shadingToWindowContent(hipSurfaceObject_t normal_map, const Window& window, const Sensor& sensor)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		shadingToWindowContentKernel << <blocks, threads >> > (normal_map, window.get_content(), sensor);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}
}