#include "hip/hip_runtime.h"
#include "measurement.cuh"
#include "device_helper.cuh"

#include <iostream>

#include <hip/hip_fp16.h>
#include <glm/vec3.hpp>
#include "cuda_event.h"

//TODO: How do we decide?
constexpr float cSigmaS = 4.0f;
constexpr float cSigmaR = 0.25f;

__global__ void convertToDepthMetersKernel(hipSurfaceObject_t raw, hipSurfaceObject_t raw_meters)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned short h_depth;
	surf2Dread(&h_depth, raw, i * 2, j);
	auto depth = __half2float(h_depth);

	//Convert depth value to value in meters
	surf2Dwrite(depth, raw_meters, i * 4, j);
}

__global__ void applyBilateralFilterKernel(hipSurfaceObject_t raw, hipSurfaceObject_t filtered)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	constexpr int w_size = 7;

	//Do not change.
	constexpr int half_w_size = w_size / 2;
	constexpr float one_over_sigmasqr_s = 1.0f / (cSigmaS * cSigmaS);
	constexpr float one_over_sigmasqr_r = 1.0f / (cSigmaR * cSigmaR);

	float center, current;
	surf2Dread(&center, raw, i * 4, j);
	auto normalization = 0.0f;
	auto acc = 0.0f;
	for (int x = -half_w_size; x <= half_w_size; ++x)
	{
		for (int y = -half_w_size; y <= half_w_size; ++y)
		{
			surf2Dread(&current, raw, (i + x) * 4, j + y, hipBoundaryModeClamp);

			auto s_dist_sqr = static_cast<float>(x * x + y * y);
			auto i_dist_sqr = (center - current);
			i_dist_sqr *= i_dist_sqr;
			auto factor = expf(-s_dist_sqr * one_over_sigmasqr_s - i_dist_sqr * one_over_sigmasqr_r);
			normalization += factor;

			acc += factor * current;
		}
	}

	surf2Dwrite(acc / normalization, filtered, i * 4, j);
}

__global__ void downSampleKernel(hipSurfaceObject_t source, hipSurfaceObject_t destination)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int idx_i = i * 8;
	int idx_j = j * 2;
	float f1, f2, f3, f4;
	surf2Dread(&f1, source, idx_i, idx_j);
	surf2Dread(&f2, source, idx_i + 4, idx_j);
	surf2Dread(&f3, source, idx_i, idx_j + 1);
	surf2Dread(&f4, source, idx_i + 4, idx_j + 1);

	constexpr float three_std_dev = 3.0f * cSigmaR;

	auto acc = f1;
	int count = 1;
	if (fabsf(f1 - f2) <= three_std_dev)
	{
		acc += f2;
		++count;
	}
	if (fabsf(f1 - f3) <= three_std_dev)
	{
		acc += f3;
		++count;
	}
	if (fabsf(f1 - f4) <= three_std_dev)
	{
		acc += f4;
		++count;
	}

	surf2Dwrite(acc / count, destination, i * 4, j);
}

__global__ void createVertexMapKernel(hipSurfaceObject_t depth_frame, hipSurfaceObject_t vertex_map, glm::mat3 inv_cam_k, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float depth;
	surf2Dread(&depth, depth_frame, i * 4, j);

	if (device_helper::isDepthValid(depth))
	{
		glm::vec3 p(i + 0.5f, j + 0.5f, 1.0f);
		p.x *= scale;
		p.y *= scale;
		p = inv_cam_k * p;
		p *= depth;

		device_helper::writeVec3(p, vertex_map, i, j);
		device_helper::validate(vertex_map, i, j);
	}
	else
	{
		device_helper::invalidate(vertex_map, i, j);
	}
}

__global__ void createNormalMapKernel(hipSurfaceObject_t vertex_map, hipSurfaceObject_t normal_map)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

    auto normal = device_helper::computeNormal(vertex_map, i, j);
	device_helper::writeVec3(normal, normal_map, i, j);
}

__global__ void oneHalfChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned short h_pixel;
	surf2Dread(&h_pixel, surface, i * 2, j);

	auto pixel = static_cast<unsigned char>(__half2float(h_pixel) * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

__global__ void oneFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float f_pixel;
	surf2Dread(&f_pixel, surface, i * 4, j);

	auto pixel = static_cast<unsigned char>(f_pixel * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

__global__ void fourFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float r, g, b;
	int idx = i * 16;
	surf2Dread(&r, surface, idx, j);
	surf2Dread(&g, surface, idx + 4, j);
	surf2Dread(&b, surface, idx + 8, j);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(b * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(g * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(r * scale));

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

namespace kernel
{
	float convertToDepthMeters(hipSurfaceObject_t input, hipSurfaceObject_t output)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		convertToDepthMetersKernel << <blocks, threads >> > (input, output);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float applyBilateralFilter(hipSurfaceObject_t input, hipSurfaceObject_t output)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		applyBilateralFilterKernel << <blocks, threads >> > (input, output);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float downSample(hipSurfaceObject_t input, hipSurfaceObject_t output, int output_width, int output_height)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(output_width / threads.x, output_height / threads.y);
		start.record();
		downSampleKernel << <blocks, threads >> > (input, output);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float createVertexMap(hipSurfaceObject_t input_depth, hipSurfaceObject_t output_vertex, const glm::mat3& inv_cam_k, int width, int height)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(width / threads.x, height / threads.y);
		start.record();
		createVertexMapKernel << <blocks, threads >> > (input_depth, output_vertex, inv_cam_k, 640 / width);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float computeNormalMap(CudaGridMap &vertex_map, CudaGridMap &normal_map)
	{
        auto dims = vertex_map.getGridDims();

		CudaEvent start, end;
		dim3 threads(8, 8);
        dim3 blocks(dims[0] / threads.x, dims[1] / threads.y);
		start.record();
		createNormalMapKernel << <blocks, threads >> > (vertex_map.getCudaSurfaceObject(), normal_map.getCudaSurfaceObject());
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float oneHalfChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneHalfChannelToWindowContentKernel << <blocks, threads >> > (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float oneFloatChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneFloatChannelToWindowContentKernel << <blocks, threads >> > (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float fourFloatChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		fourFloatChannelToWindowContentKernel <<<blocks, threads >>> (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}
}