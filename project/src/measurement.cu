#include "hip/hip_runtime.h"
#include "measurement.cuh"

#include <iostream>

#include <hip/hip_fp16.h>
#include <glm/vec3.hpp>
#include "cuda_event.h"

//TODO: How do we decide?
constexpr float cSigmaS = 4.0f;
constexpr float cSigmaR = 0.25f;

__global__ void applyBilateralFilterKernel(hipSurfaceObject_t raw, hipSurfaceObject_t filtered)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	constexpr int w_size = 7;

	//Do not change.
	constexpr int half_w_size = w_size / 2;
	constexpr float one_over_sigmasqr_s = 1.0f / (cSigmaS * cSigmaS);
	constexpr float one_over_sigmasqr_r = 1.0f / (cSigmaR * cSigmaR);

	unsigned short h_center, h_current;
	surf2Dread(&h_center, raw, i * 2, j);
	auto center = __half2float(h_center);
	auto normalization = 0.0f;
	auto acc = 0.0f;
	for (int x = -half_w_size; x <= half_w_size; ++x)
	{
		for (int y = -half_w_size; y <= half_w_size; ++y)
		{
			surf2Dread(&h_current, raw, (i + x) * 2, j + y, hipBoundaryModeClamp);
			auto current = __half2float(h_current);

			auto s_dist_sqr = static_cast<float>(x * x + y * y);
			auto i_dist_sqr = (center - current);
			i_dist_sqr *= i_dist_sqr;
			auto factor = expf(-s_dist_sqr * one_over_sigmasqr_s - i_dist_sqr * one_over_sigmasqr_r);
			normalization += factor;

			acc += factor * current;
		}
	}

	surf2Dwrite(acc / normalization, filtered, i * 4, j);
}

__global__ void downSampleKernel(hipSurfaceObject_t source, hipSurfaceObject_t destination)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	//Just average for now. Do the same as in the paper later on.
	int idx_i = i * 8;
	int idx_j = j * 2;
	float f1, f2, f3, f4;
	surf2Dread(&f1, source, idx_i, idx_j);
	surf2Dread(&f2, source, idx_i + 4, idx_j);
	surf2Dread(&f3, source, idx_i, idx_j + 1);
	surf2Dread(&f4, source, idx_i + 4, idx_j + 1);

	constexpr float three_std_dev = 3.0f * cSigmaR;

	auto acc = f1;
	int count = 1;
	if (fabsf(f1 - f2) <= three_std_dev)
	{
		acc += f2;
		++count;
	}
	if (fabsf(f1 - f3) <= three_std_dev)
	{
		acc += f3;
		++count;
	}
	if (fabsf(f1 - f4) <= three_std_dev)
	{
		acc += f4;
		++count;
	}

	surf2Dwrite(acc / count, destination, i * 4, j);
}

__global__ void createVertexMapKernel(hipSurfaceObject_t depth_frame, hipSurfaceObject_t vertex_map, glm::mat3 inv_cam_k, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float depth;
	surf2Dread(&depth, depth_frame, i * 4, j);

	glm::vec3 p(i + 0.5f, j + 0.5f, 1.0f);
	p.x *= scale;
	p.y *= scale;
	p = inv_cam_k * p;
	p *= depth;

	int idx = i * 16;
	surf2Dwrite(p.x, vertex_map, idx, j);
	surf2Dwrite(p.y, vertex_map, idx + 4, j);
	surf2Dwrite(p.z, vertex_map, idx + 8, j);
}

__global__ void createNormalMapKernel(hipSurfaceObject_t vertex_map, hipSurfaceObject_t normal_map)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;


    glm::vec3 normal = computeNormal(vertex_map, i, j);
	int idx = i * 16;
	surf2Dwrite(normal.x, normal_map, idx, j);
	surf2Dwrite(normal.y, normal_map, idx + 4, j);
	surf2Dwrite(normal.z, normal_map, idx + 8, j);
}

__global__ void oneHalfChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned short h_pixel;
	surf2Dread(&h_pixel, surface, i * 2, j);

	auto pixel = static_cast<unsigned char>(__half2float(h_pixel) * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

__global__ void oneFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float f_pixel;
	surf2Dread(&f_pixel, surface, i * 4, j);

	auto pixel = static_cast<unsigned char>(f_pixel * scale);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel) << 8;
	pixel_w = (pixel_w | pixel);

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

__global__ void fourFloatChannelToWindowContentKernel(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	float r, g, b;
	int idx = i * 16;
	surf2Dread(&r, surface, idx, j);
	surf2Dread(&g, surface, idx + 4, j);
	surf2Dread(&b, surface, idx + 8, j);

	unsigned int pixel_w = (255) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(b * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(g * scale)) << 8;
	pixel_w = (pixel_w | static_cast<unsigned char>(r * scale));

	surf2Dwrite(pixel_w, window_content, i * 4, j);
}

__device__ glm::vec3 computeNormal(hipSurfaceObject_t vertex_map, unsigned int u, unsigned int v)
{
    glm::vec3 central_vertex, next_in_row, next_in_column;
    int idx = u * 16;
    surf2Dread(&central_vertex.x, vertex_map, idx, v);
    surf2Dread(&central_vertex.y, vertex_map, idx + 4, v);
    surf2Dread(&central_vertex.z, vertex_map, idx + 8, v);

    surf2Dread(&next_in_row.x, vertex_map, idx + 16, v, hipBoundaryModeClamp);
    surf2Dread(&next_in_row.y, vertex_map, idx + 20, v, hipBoundaryModeClamp);
    surf2Dread(&next_in_row.z, vertex_map, idx + 24, v, hipBoundaryModeClamp);

    surf2Dread(&next_in_column.x, vertex_map, idx, v + 1, hipBoundaryModeClamp);
    surf2Dread(&next_in_column.y, vertex_map, idx + 4, v + 1, hipBoundaryModeClamp);
    surf2Dread(&next_in_column.z, vertex_map, idx + 8, v + 1, hipBoundaryModeClamp);

    return glm::normalize(glm::cross(next_in_row - central_vertex, next_in_column - central_vertex));
}

namespace kernel
{
	float applyBilateralFilter(hipSurfaceObject_t input, hipSurfaceObject_t output)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		applyBilateralFilterKernel << <blocks, threads >> > (input, output);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float downSample(hipSurfaceObject_t input, hipSurfaceObject_t output, int output_width, int output_height)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(output_width / threads.x, output_height / threads.y);
		start.record();
		downSampleKernel << <blocks, threads >> > (input, output);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float createVertexMap(hipSurfaceObject_t input_depth, hipSurfaceObject_t output_vertex, const glm::mat3& inv_cam_k, int width, int height)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(width / threads.x, height / threads.y);
		start.record();
		createVertexMapKernel << <blocks, threads >> > (input_depth, output_vertex, inv_cam_k, 640 / width);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float computeNormalMap(CudaGridMap &vertex_map, CudaGridMap &normal_map)
	{
        auto dims = vertex_map.getGridDims();

		CudaEvent start, end;
		dim3 threads(8, 8);
        dim3 blocks(dims[0] / threads.x, dims[1] / threads.y);
		start.record();
		createNormalMapKernel << <blocks, threads >> > (vertex_map.getCudaSurfaceObject(), normal_map.getCudaSurfaceObject());
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float oneHalfChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneHalfChannelToWindowContentKernel << <blocks, threads >> > (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float oneFloatChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		oneFloatChannelToWindowContentKernel << <blocks, threads >> > (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}

	float fourFloatChannelToWindowContent(hipSurfaceObject_t surface, hipSurfaceObject_t window_content, float scale)
	{
		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(640 / threads.x, 480 / threads.y);
		start.record();
		fourFloatChannelToWindowContentKernel <<<blocks, threads >>> (surface, window_content, scale);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}
}