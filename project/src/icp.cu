#include "hip/hip_runtime.h"
#include "icp.cuh"

#include "device_helper.cuh"

__global__ void constructIcpResidualsKernel(hipSurfaceObject_t vertex_map, hipSurfaceObject_t target_vertex_map, 
    hipSurfaceObject_t &target_normal_map, glm::mat3x3 &prev_rot_mat, glm::vec3 &prev_transl_vec, 
    glm::mat3x3 &curr_rot_mat_estimate, glm::vec3 current_transl_vec_estimate, glm::mat3x3 &sensor_intrinsics, 
    unsigned int width, unsigned int height, float distance_thresh, float angle_thresh, float mat_A[][6], float vec_b[])
{
    /* TODO:
        1. Compute indices (u, v) from kernel identifier
        2. Check using device_helpers::is_valid() whether the vertex is valid, else writeDummyResidual() and return
        3. Transform the vertex into the global frame using the previous pose
        4. Run computeCorrespondence() to get the coordinates of the corresponding element of the target vertex map
        5. Check for the distance constraint using verticesAreTooFarAway(), else writeDummyResidual() and return
        6. Compute normals for both, the current vertex map and the predicted vertex map
        7. Check for the angle constraint using normalsAreTooDifferent(), else writeDummyResidual() and return
        8. Compute the parameters for A and write them into the array using computeAndFillA()
        9. Compute the value for b and write it into the array using computeAndFillB()
    */
}

namespace kernel
{
    float constructIcpResiduals(CudaGridMap vertex_map, CudaGridMap target_vertex_map, CudaGridMap target_normal_map, 
        RigidTransform3D & previous_pose, RigidTransform3D current_pose_estimate, glm::mat3x3 & sensor_intrinsics, 
        float distance_thresh, float angle_thresh, float mat_A[][6], float vec_b[])
    {
        return 0.0f;
    }
}

__device__ std::array<int, 2> computeCorrespondence(glm::vec3 &vertex_global, glm::mat3x3 &prev_rot_mat, 
    glm::vec3 &prev_transl_vec, glm::mat3x3 &sensor_intrinsics)
{
    // TODO: Implement

	auto point = sensor_intrinsics*glm::inverse(prev_rot_mat)*(vertex_global - prev_transl_vec);

	std::array<int, 2>{ { point.x/point.z, point.y/point.z } };
}

__device__ void writeDummyResidual(float vec_a[], float *scalar_b) 
{
	*scalar_b = 0.0f;
	for (int i = 0; i < 6; i++)
		vec_a[i] = 0.0f;
}

__device__ bool verticesAreTooFarAway(glm::vec3 &vertex_1, glm::vec3 &vertex_2, float distance_thresh) 
{
    return glm::distance(vertex_1, vertex_2) > distance_thresh;
}

//TODO: couldn't find an angle function in glm, should check again
__device__ bool normalsAreTooDifferent(glm::vec3 &normal, glm::vec3 &target_normal, glm::mat3x3 &rotation_mat,
	float angle_thresh) 
{
	glm::vec3 new_normal = normal * rotation_mat;
	glm::vec3 da = glm::normalize(new_normal);
	glm::vec3 db = glm::normalize(target_normal);
	float angle= glm::acos(glm::dot(da, db));
	
    return angle > angle_thresh;
}

__device__ void computeAndFillA(float vec_a[], glm::vec3 &vertex_global, glm::vec3 &target_normal) 
{
	const auto& s = vertex_global;
	const auto& n = target_normal;
	vec_a[0] = n.y*s.z - n.z*s.y;
	vec_a[1] = n.z*s.x - n.x*s.z;
	vec_a[2] = n.x*s.y - n.y*s.x;
	vec_a[3] = n.x;
	vec_a[4] = n.y;
	vec_a[5] = n.z;

}

__device__ void computeAndFillB(float *scalar_b, glm::vec3 &vertex_global, glm::vec3 &target_vertex,
glm::vec3 &target_normal) 
{
	const auto& s = vertex_global;
	const auto& n = target_normal;
	const auto& d = target_vertex;
	*scalar_b = n.x*d.x + n.y*d.y + n.z*d.z - n.x*s.x - n.y*s.y - n.z*s.z;
}