#include "hip/hip_runtime.h"
#include "tsdf.cuh"
#include "cuda_event.h"
#include "device_helper.cuh"

//If there is an intersection between the bbox and ray, parameters (t) of close and far intersection points are returned.
//From https://github.com/isikmustafa/pathtracer/blob/master/bbox.cuh
__device__ glm::vec2 intersectBBox(const glm::vec3& origin, const glm::vec3& inv_dir, const glm::vec3& min, const glm::vec3& max)
{
	auto t0 = (min - origin) * inv_dir;
	auto t1 = (max - origin) * inv_dir;

	auto tmin = fminf(t0.x, t1.x);
	auto tmax = fmaxf(t0.x, t1.x);

	tmin = fmaxf(tmin, fminf(t0.y, t1.y));
	tmax = fminf(tmax, fmaxf(t0.y, t1.y));

	tmin = fmaxf(tmin, fminf(t0.z, t1.z));
	tmax = fminf(tmax, fmaxf(t0.z, t1.z));

	if (tmax < tmin)
	{
		return glm::vec2(-1.0f, -1.0f);
	}

	return glm::vec2(tmin, tmax);
}

__device__ glm::vec3 computeGradient(const glm::vec3& point, const VoxelGridStruct& voxel_grid)
{
	auto uvw_resolution = 0.5f / (voxel_grid.n - 1);
	auto uvw = point / voxel_grid.total_width_in_meters + glm::vec3(0.5f);

	auto f_x0 = tex3D<float2>(voxel_grid.texture_object, uvw.x + uvw_resolution, uvw.y, uvw.z).x;
	auto f_y0 = tex3D<float2>(voxel_grid.texture_object, uvw.x, uvw.y + uvw_resolution, uvw.z).x;
	auto f_z0 = tex3D<float2>(voxel_grid.texture_object, uvw.x, uvw.y, uvw.z + uvw_resolution).x;
	auto f_x = tex3D<float2>(voxel_grid.texture_object, uvw.x - uvw_resolution, uvw.y, uvw.z).x;
	auto f_y = tex3D<float2>(voxel_grid.texture_object, uvw.x, uvw.y - uvw_resolution, uvw.z).x;
	auto f_z = tex3D<float2>(voxel_grid.texture_object, uvw.x, uvw.y, uvw.z - uvw_resolution).x;

	return (glm::vec3(f_x, f_y, f_z) - glm::vec3(f_x0, f_y0, f_z0)) / uvw_resolution;
}

__global__ void raycastKernel(VoxelGridStruct voxel_grid, Sensor sensor, hipSurfaceObject_t output_vertex, hipSurfaceObject_t output_normal, int level)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	const auto mue = voxel_grid.mue;

	auto ray_origin = sensor.getPosition();
	//Do not normalize the direction. pos = origin + dir * depth.
	auto ray_direction = glm::mat3(sensor.getPose()) * sensor.getInverseIntr(level) * glm::vec3(i + 0.5f, j + 0.5f, 1.0f);

	//For an efficient and correct solution, intersect the ray first with bounding box of the voxel grid to determine near and far distance
	//for ray casting.
	auto half_total_width = voxel_grid.total_width_in_meters * 0.5f;
	auto result = intersectBBox(ray_origin, 1.0f / ray_direction, glm::vec3(-half_total_width), glm::vec3(half_total_width));

	//If voxel grid is behind the camera or has no intersection with this ray.
	if (result.y < 0.0f)
	{
		return;
	}

	auto near_distance = glm::max(kernel::cMinDistance, result.x);
	auto far_distance = glm::min(kernel::cMaxDistance, result.y);

	//If view frustum does not intersect voxel grid from neither sides.
	if (near_distance >= far_distance)
	{
		return;
	}

	auto distance_increase = mue * 0.99f;
	auto previous_tsdf = 0.0f;
	auto precise_distance = 0.0f;
	for (auto current_distance = near_distance; current_distance < far_distance; current_distance += distance_increase)
	{
		//1-Find the current point on the ray.
		auto current_point = ray_origin + ray_direction * current_distance;

		//2-Find trilinearly interpolated TSDF value of the current_point.
		auto uvw = current_point / voxel_grid.total_width_in_meters + glm::vec3(0.5f);
		auto tsdf = tex3D<float2>(voxel_grid.texture_object, uvw.x, uvw.y, uvw.z).x;

		//3-Check TSDF value of the voxel and determine if this is a zero crossing or not.
		if (tsdf < 0.0f)
		{
			//If this is not the first iteration, it means the ray does not intersect a backfacing surface
			//and continued from +ve to -ve.
			if (current_distance != near_distance)
			{
				//Formula(15) to compute more precise distance of intersection.
				precise_distance = current_distance - distance_increase * previous_tsdf / (tsdf - previous_tsdf);
			}
			break;
		}
		//4-Update distance_increase if it is the region of uncertainty.
		else if (tsdf < 0.99f)
		{
			distance_increase = mue * 0.125f;
		}
		previous_tsdf = tsdf;
	}

	//Ray intersected a surface and no backfacing surface is found.
	if (precise_distance != 0.0f)
	{
		auto vertex = ray_origin + ray_direction * precise_distance;
		auto normal = computeGradient(vertex, voxel_grid);
		auto normal_norm = glm::length(normal);

		if (device_helper::isDepthValid(normal_norm))
		{
			//Write vertex.
			device_helper::writeVec3(vertex, output_vertex, i, j);
			device_helper::validate(output_vertex, i, j);

			//Write normal.
			normal /= normal_norm;
			device_helper::writeVec3(normal, output_normal, i, j);
            device_helper::validate(output_normal, i, j);

			return;
		}
	}

	device_helper::writeVec3(glm::vec3(0.0f), output_vertex, i, j);
	device_helper::invalidate(output_vertex, i, j);
	device_helper::writeVec3(glm::vec3(0.0f), output_normal, i, j);
    device_helper::invalidate(output_normal, i, j);
}

namespace kernel
{
	float raycast(const VoxelGridStruct& voxel_grid, const Sensor& sensor, const CudaGridMap& output_vertex, const CudaGridMap& output_normal, int level)
	{
		auto dims_vertex = output_vertex.getGridDims();
		auto dims_normal = output_normal.getGridDims();

		if (dims_vertex != dims_normal)
		{
			throw std::runtime_error("raycast: vertex and normal surface objects are not of same size!");
		}

		CudaEvent start, end;
		dim3 threads(8, 8);
		dim3 blocks(dims_vertex[0] / threads.x, dims_vertex[1] / threads.y);
		start.record();
		raycastKernel << <blocks, threads >> > (voxel_grid, sensor, output_vertex.getCudaSurfaceObject(), output_normal.getCudaSurfaceObject(), level);
		end.record();
		end.synchronize();

		return CudaEvent::calculateElapsedTime(start, end);
	}
}