#include "hip/hip_runtime.h"
#include "cuda_wrapper.cuh"

__global__ void computeCorrespondenceTestKernel(std::array<int, 2> *result_coords, glm::vec3 vertex_global, 
    glm::mat3x3 rotation_mat, glm::vec3 translation_vec, glm::mat3x3 sensor_intrinsics)
{
    *result_coords = computeCorrespondence(vertex_global, rotation_mat, translation_vec, sensor_intrinsics);
}

std::array<int, 2> computeCorrespondenceTestWrapper(glm::vec3 vertex, glm::mat3x3 rotation_mat, 
    glm::vec3 translation_vec, glm::mat3x3 intrinsics)
{
    std::array<int, 2> *result_device;
    HANDLE_ERROR(hipMalloc(&result_device, sizeof(std::array<int, 2>)));

    computeCorrespondenceTestKernel<<<1, 1>>> (result_device, vertex, rotation_mat, translation_vec, intrinsics);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    std::array<int, 2> result_host;
    HANDLE_ERROR(hipMemcpy(&result_host, result_device, sizeof(std::array<int, 2>), hipMemcpyDeviceToHost));

    return result_host;
}


__global__ void cumputeNormalTestKernel(glm::vec3 *normal, hipSurfaceObject_t vertices, unsigned int u, unsigned int v)
{
    *normal = device_helper::computeNormal(vertices, u, v);
}

glm::vec3 computeNormalTestWrapper(CudaGridMap &vertex_map, unsigned int u, unsigned int v)
{
    glm::vec3 *normal_device;
    int size = sizeof(glm::vec3);
    HANDLE_ERROR(hipMalloc(&normal_device, size));

    cumputeNormalTestKernel<<<1, 1>>>(normal_device, vertex_map.getCudaSurfaceObject(), u, v);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    glm::vec3 normal_host;
    HANDLE_ERROR(hipMemcpy(&normal_host, normal_device, size, hipMemcpyDeviceToHost));
    
    return normal_host;

}


__global__ void normalsAreTooDifferentTestKernel(bool *result, glm::vec3 normal, glm::vec3 target_normal, 
    glm::mat3x3 rotation_mat, float angle_thresh)
{
    *result = normalsAreTooDifferent(normal, target_normal, rotation_mat, angle_thresh);
}

bool normalsAreTooDifferentTestWrapper(glm::vec3 normal, glm::vec3 target_normal, glm::mat3x3 rotation_mat, 
    float angle_thresh)
{
    bool *result_device;
    HANDLE_ERROR(hipMalloc(&result_device, sizeof(bool)));

    normalsAreTooDifferentTestKernel<<<1, 1>>>(result_device, normal, target_normal, rotation_mat, angle_thresh);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    bool result_host;
    HANDLE_ERROR(hipMemcpy(&result_host, result_device, sizeof(bool), hipMemcpyDeviceToHost));

    return result_host;
}


__global__ void computeAndFillATestKernel(float *mat_a, glm::vec3 vertex, glm::vec3 normal)
{
    computeAndFillA(mat_a, vertex, normal);
}

void computeAndFillATestWrapper(std::array<float, 6> *result, glm::vec3 vertex, glm::vec3 normal)
{
    float *result_device;
    HANDLE_ERROR(hipMalloc(&result_device, sizeof(float) * 6));

    computeAndFillATestKernel<<<1, 1>>>(result_device, vertex, normal);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(result, result_device, sizeof(float) * 6, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipDeviceSynchronize());
}


__global__ void computeAndFillBTestKernel(float *scalar_b, glm::vec3 vertex, glm::vec3 target_vertex, 
    glm::vec3 target_normal)
{
    computeAndFillB(scalar_b, vertex, target_vertex, target_normal);
}

float computeAndFillBTestWrapper(glm::vec3 vertex, glm::vec3 target_vertex, glm::vec3 target_normal)
{
    float *result_device;
    HANDLE_ERROR(hipMalloc(&result_device, sizeof(float)));

    computeAndFillBTestKernel<<<1, 1>>> (result_device, vertex, target_vertex, target_normal);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    float result_host;
    HANDLE_ERROR(hipMemcpy(&result_host, result_device, sizeof(float), hipMemcpyDeviceToHost));
    return result_host;
}
